#define GOOGLE_CUDA 1
#define EIGEN_USE_GPU

#include "tensorflow/core/framework/register_types.h"
#include "tensorflow/core/framework/tensor_types.h"
#include "tensorflow/core/util/gpu_kernel_helper.h"

// See https://kailaix.github.io/ADCME.jl/dev/customop/#GPU-Operators-1 for implementation details

namespace tensorflow{
  typedef Eigen::GpuDevice GPUDevice;

  
}